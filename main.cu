#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include "./src/library.h"

using namespace std;


int WIDTH;
__device__ int d_WIDTH;

int xSobel[3][3] = {{1,0,-1},{2,0,-2},{1,0,-1}};
int ySobel[3][3] = {{1,2,1},{0,0,0},{-1,-2,-1}};
__constant__ int d_xSobel[9] = {1, 0, -1, 2, 0, -2, 1, 0, -1};
__constant__ int d_ySobel[9] = {1, 2, 1, 0, 0, 0, -1, -2, -1};
const int filterWidth = 3;


/**
 * @param argc[1] name of the input file (.pmn)
 * @param argc[2] name of output file with no extension, created by using host & device
 * @param argc[3] horizontal of image you want to resize 
 * @param argc[4] - optional - default(32): blocksize.x
 * @param argc[5] - optional - default(32): blocksize.y
 */
void checkInput(int argc, char ** argv, int &width, int &height, uchar3 * &rgbPic, int &desiredWidth, dim3 &blockSize) {
    if (argc != 4 && argc != 6) {
        printf("The number of arguments is invalid\n");
        exit(EXIT_FAILURE);
    }

    // Read file
    readPnm(argv[1], width, height, rgbPic);
    printf("Image size (width x height): %i x %i\n\n", width, height);

    WIDTH = width;
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_WIDTH), &width, sizeof(int)));

    // Check user's desired width
    desiredWidth = atoi(argv[3]);

    if (desiredWidth <= 0 || desiredWidth >= width) {
        printf("Your desired width must between 0 & current picture's width!\n");
        exit(EXIT_FAILURE);
    }

    // Block size
    if (argc == 6) {
        blockSize.x = atoi(argv[4]);
        blockSize.y = atoi(argv[5]);
    } 

    // Check GPU is working or not
    printDeviceInfo();
}


__global__ void convertRgb2GrayKernel(uchar3 * rgbPic, int width, int height, uint8_t * grayPic) {
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;

    if (r < height && c < width) {
        int i = r * width + c;
        grayPic[i] = 0.299f*rgbPic[i].x + 0.587f*rgbPic[i].y + 0.114f*rgbPic[i].z;
    }
}


__global__ void calEnergy(uint8_t * inPixels, int width, int height, int * energy) {

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int s_width = blockDim.x + filterWidth - 1;
    int s_height = blockDim.y + filterWidth - 1;

    // Each block loads data from GMEM to SMEM
    extern __shared__ uint8_t s_inPixels[];

    int readRow = row - filterWidth / 2, readCol, tmpRow, tmpCol;
    int firstReadCol = col - filterWidth / 2;
    int virtualRow, virtualCol;

    for (virtualRow = threadIdx.y; virtualRow < s_height; readRow += blockDim.y, virtualRow += blockDim.y) {
        tmpRow = readRow;

        // if (readRow < 0)
        //     readRow = 0;
        // else if (readRow >= height) 
        //     readRow = height - 1;

        readRow = min(max(readRow, 0), height - 1);//0 <= readCol <= height-1
        
        readCol = firstReadCol;
        virtualCol = threadIdx.x;

        for (; virtualCol < s_width; readCol += blockDim.x, virtualCol += blockDim.x) {
            tmpCol = readCol;

            // if (readCol < 0) 
            //     readCol = 0;
            // else if (readCol >= width) 
            //     readCol = width - 1;

            readCol = min(max(readCol, 0), width - 1);// 0 <= readCol <= width-1
            
            s_inPixels[virtualRow * s_width + virtualCol] = inPixels[readRow * d_WIDTH + readCol];
            readCol = tmpCol;
        }
        readRow = tmpRow;
    } 
    __syncthreads();


    // Each thread compute energy on SMEM
    int x_kernel = 0, y_kernel = 0;
    for (int i = 0; i < filterWidth; ++i) {
        for (int j = 0; j < filterWidth; ++j) {
            uint8_t closest = s_inPixels[(threadIdx.y + i) * s_width + threadIdx.x + j];
            int filterIdx = i * filterWidth + j;
            x_kernel += closest * d_xSobel[filterIdx];
            y_kernel += closest * d_ySobel[filterIdx];
        }
    }

    // Each thread writes result from SMEM to GMEM
    if (col < width && row < height)
        energy[row * d_WIDTH + col] = abs(x_kernel) + abs(y_kernel);
}

__global__ void carvingKernel(int * leastSignificantPixel, uchar3 * outPixels, uint8_t *grayPixels, int * energy, int width) {
    int row = blockIdx.x;
    int baseIdx = row * d_WIDTH;
    for (int i = leastSignificantPixel[row]; i < width - 1; ++i) {
        outPixels[baseIdx + i] = outPixels[baseIdx + i + 1];
        grayPixels[baseIdx + i] = grayPixels[baseIdx + i + 1];
        energy[baseIdx + i] = energy[baseIdx + i + 1];
    }
}

void findSeam(int * minimalEnergy, int *leastSignificantPixel, int width, int height) {
    int minCol = 0, r = height - 1;

    for (int c = 1; c < width; ++c)
        if (minimalEnergy[r * WIDTH + c] < minimalEnergy[r * WIDTH + minCol])
            minCol = c;
    
    for (; r >= 0; --r) {
        leastSignificantPixel[r] = minCol;
        if (r > 0) {
            int aboveIdx = (r - 1) * WIDTH + minCol;
            int min = minimalEnergy[aboveIdx], minColCpy = minCol;

            if (minColCpy > 0 && minimalEnergy[aboveIdx - 1] < min) {
                min = minimalEnergy[aboveIdx - 1];
                minCol = minColCpy - 1;
            }
            if (minColCpy < width - 1 && minimalEnergy[aboveIdx + 1] < min) {
                minCol = minColCpy + 1;
            }
        }
    }
}

__global__ void energyToTheEndKernel(int * energy, int * minimalEnergy, int width, int height, int fromRow) {
    size_t halfBlock = blockDim.x / 2;//blockDim.x >> 1

    int col = blockIdx.x * halfBlock - halfBlock + threadIdx.x;

    if (fromRow == 0 && col >= 0 && col < width) {
        minimalEnergy[col] = energy[col];
    }
    __syncthreads();

    for (int stride = fromRow != 0 ? 0 : 1; stride < halfBlock && fromRow + stride < height; ++stride) {
        if (threadIdx.x < blockDim.x - (stride << 1)) {
            int curRow = fromRow + stride;
            int curCol = col + stride;

            if (curCol >= 0 && curCol < width) {
                int idx = curRow * d_WIDTH + curCol;
                int aboveIdx = (curRow - 1) * d_WIDTH + curCol;

                int min = minimalEnergy[aboveIdx];
                if (curCol > 0 && minimalEnergy[aboveIdx - 1] < min)
                    min = minimalEnergy[aboveIdx - 1];
                
                if (curCol < width - 1 && minimalEnergy[aboveIdx + 1] < min)
                    min = minimalEnergy[aboveIdx + 1];
                

                minimalEnergy[idx] = min + energy[idx];
            }
        }
        __syncthreads();
    }
}

void deviceResizing(uchar3 * inPixels, int width, int height, int desiredWidth, uchar3 * outPixels, dim3 blockSize) {
    GpuTimer timer;
    timer.Start();

    // allocate kernel memory
    uchar3 * d_inPixels;
    CHECK(hipMalloc(&d_inPixels, width * height * sizeof(uchar3)));
    uint8_t * d_grayPixels;
    CHECK(hipMalloc(&d_grayPixels, width * height * sizeof(uint8_t)));
    int * d_energy;
    CHECK(hipMalloc(&d_energy, width * height * sizeof(int)));
    int * d_leastSignificantPixel;
    CHECK(hipMalloc(&d_leastSignificantPixel, height * sizeof(int)));
    int * d_minimalEnergy;
    CHECK(hipMalloc(&d_minimalEnergy, width * height * sizeof(int)));

    // allocate host memory
    int * energy = (int *)malloc(width * height * sizeof(int));
    int * leastSignificantPixel = (int *)malloc(height * sizeof(int));
    int * minimalEnergy = (int *)malloc(width * height * sizeof(int));

    // dynamically sized smem used to compute energy
    size_t smemSize = ((blockSize.x + 3 - 1) * (blockSize.y + 3 - 1)) * sizeof(uint8_t);
    
    // block size use to calculate minimal energy to the end
    int blockSizeDp = 256;
    int gridSizeDp = (((width - 1) / blockSizeDp + 1) << 1) + 1;
    int stripHeight = (blockSizeDp >> 1) + 1;

    // copy input to device
    CHECK(hipMemcpy(d_inPixels, inPixels, width * height * sizeof(uchar3), hipMemcpyHostToDevice));

    // turn input image to grayscale
    dim3 gridSize((width-1)/blockSize.x + 1, (height-1)/blockSize.y + 1);
    convertRgb2GrayKernel<<<gridSize, blockSize>>>(d_inPixels, width, height, d_grayPixels);
    hipDeviceSynchronize();
    CHECK(hipGetLastError());

    while (width > desiredWidth) {
        // update energy
        calEnergy<<<gridSize, blockSize, smemSize>>>(d_grayPixels, width, height, d_energy);
        hipDeviceSynchronize();
        CHECK(hipGetLastError());

        // compute min seam table
        for (int i = 0; i < height; i += (stripHeight >> 1)) {
            energyToTheEndKernel<<<gridSizeDp, blockSizeDp>>>(d_energy, d_minimalEnergy, width, height, i);
            hipDeviceSynchronize();
            CHECK(hipGetLastError());
        }

        // find least significant pixel index of each row and store in d_leastSignificantPixel (SEQUENTIAL, in kernel or host)
        CHECK(hipMemcpy(minimalEnergy, d_minimalEnergy, WIDTH * height * sizeof(int), hipMemcpyDeviceToHost));
        findSeam(minimalEnergy, leastSignificantPixel, width, height);

        // carve
        CHECK(hipMemcpy(d_leastSignificantPixel, leastSignificantPixel, height * sizeof(int), hipMemcpyHostToDevice));
        carvingKernel<<<height, 1>>>(d_leastSignificantPixel, d_inPixels, d_grayPixels, d_energy, width);
        hipDeviceSynchronize();
        CHECK(hipGetLastError());
        
        --width;
    }

    CHECK(hipMemcpy(outPixels, d_inPixels, WIDTH * height * sizeof(uchar3), hipMemcpyDeviceToHost));

    CHECK(hipFree(d_inPixels));
    CHECK(hipFree(d_grayPixels));
    CHECK(hipFree(d_energy));
    CHECK(hipFree(d_leastSignificantPixel));
    CHECK(hipFree(d_minimalEnergy));

    free(minimalEnergy);
    free(leastSignificantPixel);
    free(energy);

    timer.Stop();
    timer.printTime((char *)"device");   
}

// HOST

int getPixelEnergy(uint8_t * grayPixels, int row, int col, int width, int height) {
    int x_kernel = 0;
    int y_kernel = 0;

    for (int i = 0; i < 3; ++i) { // 3: filter width
        for (int j = 0; j < 3; ++j) {
            int r = min(max(0, row - 1 + i), height - 1); // 0 <= row - 1 + i < height
            int c = min(max(0, col - 1 + j), width - 1); // 0 <= col - 1 + j < width

            uint8_t pixelVal = grayPixels[r * WIDTH + c];//

            x_kernel += pixelVal * xSobel[i][j];// Convolution with x-Sobel
            y_kernel += pixelVal * ySobel[i][j];// Convolution with y-Sobel
        }
    }
    return abs(x_kernel) + abs(y_kernel);// Add matrix
}



void energyToTheEnd(int * energy, int * minimalEnergy, int width, int height) {
    for (int c = 0; c < width; ++c) {
        minimalEnergy[c] = energy[c];
    }
    for (int r = 1; r < height; ++r) {
        for (int c = 0; c < width; ++c) {
            int idx = r * WIDTH + c;
            int aboveIdx = (r - 1) * WIDTH + c;

            int min = minimalEnergy[aboveIdx];
            if (c > 0 && minimalEnergy[aboveIdx - 1] < min) {
                min = minimalEnergy[aboveIdx - 1];
            }
            if (c < width - 1 && minimalEnergy[aboveIdx + 1] < min) {
                min = minimalEnergy[aboveIdx + 1];
            }

            minimalEnergy[idx] = min + energy[idx];
        }
    }
}

void hostResizing(uchar3 * inPixels, int width, int height, int desiredWidth, uchar3 * outPixels) {
    GpuTimer timer;
    timer.Start();

    memcpy(outPixels, inPixels, width * height * sizeof(uchar3));

    // Allocating memory
    int * energy = (int *)malloc(width * height * sizeof(int));
    int * minimalEnergy = (int *)malloc(width * height * sizeof(int));
    
    // Get grayscale
    uint8_t * grayPixels = (uint8_t *)malloc(width * height * sizeof(uint8_t));
    convertRgb2Gray_host(inPixels, width, height, grayPixels);

    // Calculate all pixels energy
    for (int r = 0; r < height; ++r) {
        for (int c = 0; c < width; ++c) {
            energy[r * WIDTH + c] = getPixelEnergy(grayPixels, r, c, width, height);
        }
    }

    while (width > desiredWidth) {
        // Calculate energy to the end. (go from bottom to top)
        energyToTheEnd(energy, minimalEnergy, width, height);

        // find min index of last row
        int minCol = 0, r = height - 1, prevMinCol;
        for (int c = 1; c < width; ++c) {
            if (minimalEnergy[r * WIDTH + c] < minimalEnergy[r * WIDTH + minCol])
                minCol = c;
        }

        // Find and remove seam from last to first row
        for (; r >= 0; --r) {
            // remove seam pixel on row r
            for (int i = minCol; i < width - 1; ++i) {
                outPixels[r * WIDTH + i] = outPixels[r * WIDTH + i + 1];
                grayPixels[r * WIDTH + i] = grayPixels[r * WIDTH + i + 1];
                energy[r * WIDTH + i] = energy[r * WIDTH + i + 1];
            }

            // Update energy
            if (r < height - 1) {
                int affectedCol = max(0, prevMinCol - 2);

                while (affectedCol <= prevMinCol + 2 && affectedCol < width - 1) {
                    energy[(r + 1) * WIDTH + affectedCol] = getPixelEnergy(grayPixels, r + 1, affectedCol, width - 1, height);
                    affectedCol += 1;
                }
            }

            // find to the top
            if (r > 0) {
                prevMinCol = minCol;

                int aboveIdx = (r - 1) * WIDTH + minCol;
                int min = minimalEnergy[aboveIdx], minColCpy = minCol;
                if (minColCpy > 0 && minimalEnergy[aboveIdx - 1] < min) {
                    min = minimalEnergy[aboveIdx - 1];
                    minCol = minColCpy - 1;
                }
                if (minColCpy < width - 1 && minimalEnergy[aboveIdx + 1] < min) {
                    minCol = minColCpy + 1;
                }
            }
        }

        int affectedCol;
        for (affectedCol=max(0, minCol - 2); affectedCol <= minCol + 2 && affectedCol < width - 1; ++affectedCol) {
            energy[affectedCol] = getPixelEnergy(grayPixels, 0, affectedCol, width - 1, height);
        }

        --width;
    }
    
    free(grayPixels);
    free(minimalEnergy);
    free(energy);

    timer.Stop();
    timer.printTime((char *)"host");
}


int main(int argc, char ** argv) {   

    int width, height, desiredWidth;
    uchar3 * rgbPic;
    dim3 blockSize(32, 32);

    // Check user's input
    checkInput(argc, argv, width, height, rgbPic, desiredWidth, blockSize);

    // HOST
    uchar3 * out_host = (uchar3 *)malloc(width * height * sizeof(uchar3));
    hostResizing(rgbPic, width, height, desiredWidth, out_host);

    // DEVICE
    uchar3 * out_device = (uchar3 *)malloc(width * height * sizeof(uchar3));
    deviceResizing(rgbPic, width, height, desiredWidth, out_device, blockSize);

    // Compute error
    printError((char * )"Error between device result and host result: ", out_host, out_device, width, height);

    // Write 2 results to files
    writePnm(out_host, desiredWidth, height, width, concatStr(argv[2], "_host.pnm"));
    writePnm(out_device, desiredWidth, height, width, concatStr(argv[2], "_device.pnm"));

    // Free memories
    free(rgbPic);
    free(out_host);
    free(out_device);
}
